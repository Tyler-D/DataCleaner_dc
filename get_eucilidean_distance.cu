#include "hip/hip_runtime.h"
#include "data_cleaner.hpp"

int cuda_check(hipError_t error){
  if (error != hipSuccess)
  {
    cout<<"CUDA error : "<<error<<endl;
    exit(0); 
  }
  else 
    return 0; 
}

int cublas_check(hipblasStatus_t stat){
  if (stat != HIPBLAS_STATUS_SUCCESS)
  {
    cout<<"CUBLAS error: "<<stat<<endl;
    exit(0);
  }
  else
    return 0;
}

__global__ void sub_kernel(int num, const float* A, const float* B, float* C)
{
  for(int index = 0; index < num; index++)
  {
    C[index] = A[index] - B[index];
  }
}

/*
__global__ void dot_kernel(int num, const float* A , const float* B, float* C)
{
    for(int index = 0; index < num; index++)
    {
        (*C)+=A[index]*B[index];
    }
}
*/

DCStatus DataCleaner:: get_Eucilidean_distance(int dim, const vector<float*>* data_pairs, float* distance){
  hipError_t error;
  hipblasStatus_t cublas_stat;
  size_t data_size = sizeof(float)*dim;

  float* dev_ptrA;
  float* dev_ptrB;
  float* dev_ptrC;
  float* dev_result;
  //set the device memory
  error = hipMalloc((void**)&dev_ptrA,data_size);
  cuda_check(error);
  error = hipMalloc((void**)&dev_ptrB,data_size);
  cuda_check(error);
  error = hipMalloc((void**)&dev_ptrC,data_size);
  cuda_check(error);
  error = hipMalloc((void**)&dev_result,sizeof(float));
  cuda_check(error);
  
  //copy the vector from host to vector
  cublas_stat = hipblasSetVector(dim, sizeof(float), (*data_pairs)[0], 1, dev_ptrA, 1);
  cublas_check(cublas_stat);
  cublas_stat = hipblasSetVector(dim, sizeof(float), (*data_pairs)[1], 1, dev_ptrB, 1);
  cublas_check(cublas_stat); 

  int threads_per_block = 256;
  int blocks_per_grid = (dim + threads_per_block - 1)/threads_per_block;
  sub_kernel<<<blocks_per_grid, threads_per_block>>>(dim, dev_ptrA,dev_ptrB,dev_ptrC);

  //dot_kernel<<<1024, threads_per_block>>>(dim, dev_ptrC, dev_ptrC, dev_result)
  cublas_stat = hipblasSdot(this->cublas_handle_, dim, dev_ptrC, 1, dev_ptrC, 1, distance); 
  cublas_check(cublas_stat);
  
  //cublas_stat = hipblasGetVector(dim, sizeof(float), dev_result, 1, distance, 1); 
  //cublas_check(cublas_stat);
  
  hipFree(dev_ptrA);
  hipFree(dev_ptrB);
  hipFree(dev_ptrC);


  return DC_SUCCESS;
}
